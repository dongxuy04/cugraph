/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <utilities/base_fixture.hpp>
#include <utilities/test_graphs.hpp>
#include <utilities/test_utilities.hpp>
#include <utilities/thrust_wrapper.hpp>

#include <cugraph/algorithms.hpp>
#include <cugraph/graph.hpp>
#include <cugraph/graph_functions.hpp>
#include <cugraph/graph_view.hpp>
#include <cugraph/mtmg/edgelist.hpp>
#include <cugraph/mtmg/graph.hpp>
#include <cugraph/mtmg/renumber_map.hpp>
#include <cugraph/mtmg/resource_manager.hpp>
#include <cugraph/mtmg/thread_edgelist.hpp>
#include <cugraph/mtmg/vertex_result.hpp>
#include <cugraph/utilities/high_res_timer.hpp>

#include <raft/util/cudart_utils.hpp>

#include <rmm/device_uvector.hpp>
#include <rmm/mr/device/cuda_memory_resource.hpp>

#include <gtest/gtest.h>

#include <nccl.h>

#include <algorithm>
#include <iterator>
#include <limits>
#include <numeric>
#include <random>
#include <vector>

#include <cugraph/utilities/device_functors.cuh>
#include <detail/graph_partition_utils.cuh>
#include <thrust/count.h>

struct Multithreaded_Usecase {
  bool test_weighted{false};
  bool check_correctness{true};
};

template <typename input_usecase_t>
class Tests_Multithreaded
  : public ::testing::TestWithParam<std::tuple<Multithreaded_Usecase, input_usecase_t>> {
 public:
  Tests_Multithreaded() {}

  static void SetUpTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}

  std::vector<int> get_gpu_list()
  {
    int num_gpus_per_node{1};
    RAFT_CUDA_TRY(hipGetDeviceCount(&num_gpus_per_node));

    std::vector<int> gpu_list(num_gpus_per_node);
    std::iota(gpu_list.begin(), gpu_list.end(), 0);

    return gpu_list;
  }

  template <typename vertex_t,
            typename edge_t,
            typename weight_t,
            typename result_t,
            bool multi_gpu>
  void run_current_test(
    std::tuple<Multithreaded_Usecase const&, input_usecase_t const&> const& param,
    std::vector<int> gpu_list)
  {
    using edge_type_t = int32_t;

    constexpr bool renumber           = true;
    constexpr bool do_expensive_check = false;

    auto [multithreaded_usecase, input_usecase] = param;

    raft::handle_t handle{};
    HighResTimer hr_timer{};

    result_t constexpr alpha{0.85};
    result_t constexpr epsilon{1e-6};

    size_t device_buffer_size{64 * 1024 * 1024};
    size_t thread_buffer_size{4 * 1024 * 1024};

    int num_gpus    = gpu_list.size();
    int num_threads = num_gpus * 4;

    cugraph::mtmg::resource_manager_t resource_manager;

    std::for_each(gpu_list.begin(), gpu_list.end(), [&resource_manager](int gpu_id) {
      resource_manager.register_local_gpu(gpu_id, rmm::cuda_device_id{gpu_id});
    });

    ncclUniqueId instance_manager_id;
    ncclGetUniqueId(&instance_manager_id);

    std::cout << "create instance_manager" << std::endl;

    auto instance_manager = resource_manager.create_instance_manager(
      resource_manager.registered_ranks(), instance_manager_id);

    cugraph::mtmg::edgelist_t<vertex_t, weight_t, edge_t, edge_type_t> edgelist;
    cugraph::mtmg::graph_t<vertex_t, edge_t, true, multi_gpu> graph;
    cugraph::mtmg::graph_view_t<vertex_t, edge_t, true, multi_gpu> graph_view;
    cugraph::mtmg::vertex_result_t<result_t> pageranks;
    std::optional<cugraph::mtmg::renumber_map_t<vertex_t>> renumber_map =
      std::make_optional<cugraph::mtmg::renumber_map_t<vertex_t>>();

    std::cout << "prepare to create edges" << std::endl;

    //
    // Simulate graph creation by spawning threads to walk through the
    // local COO and add edges
    //
    std::vector<std::thread> running_threads;

    //  Initialize shared edgelist object, one per GPU
    for (int i = 0; i < num_gpus; ++i) {
      running_threads.emplace_back([&instance_manager,
                                    &edgelist,
                                    &renumber_map,
                                    i,
                                    num_gpus,
                                    device_buffer_size,
                                    use_weight    = true,
                                    use_edge_id   = false,
                                    use_edge_type = false]() {
        auto thread_handle = instance_manager->get_handle();

        edgelist.initialize_pointer(
          thread_handle, thread_handle, device_buffer_size, use_weight, use_edge_id, use_edge_type);
        if (renumber_map)
          renumber_map->initialize_pointer(
            thread_handle, 0, thread_handle.raft_handle().get_stream());
      });
    }

    // Wait for CPU threads to complete
    std::for_each(running_threads.begin(), running_threads.end(), [](auto& t) { t.join(); });
    running_threads.resize(0);
    instance_manager->reset_threads();

    std::cout << "load sg edge list" << std::endl;

    // Load SG edge list
    auto [d_src_v, d_dst_v, d_weights_v, d_vertices_v, is_symmetric] =
      input_usecase.template construct_edgelist<vertex_t, weight_t>(
        handle, multithreaded_usecase.test_weighted, false, false);

    auto h_src_v     = cugraph::test::to_host(handle, d_src_v);
    auto h_dst_v     = cugraph::test::to_host(handle, d_dst_v);
    auto h_weights_v = cugraph::test::to_host(handle, d_weights_v);

    std::cout << "load edgelist" << std::endl;

    // Load edgelist from different threads.  We'll use more threads than GPUs here
    for (int i = 0; i < num_threads; ++i) {
      running_threads.emplace_back([&instance_manager,
                                    thread_buffer_size,
                                    &edgelist,
                                    &h_src_v,
                                    &h_dst_v,
                                    &h_weights_v,
                                    i,
                                    num_threads]() {
        sleep(i);
        auto thread_handle = instance_manager->get_handle();
        cugraph::mtmg::thread_edgelist_t<vertex_t, weight_t, edge_t, edge_type_t> thread_edgelist(
          *edgelist.get_pointer(thread_handle), thread_buffer_size);

        for (int j = i; j < h_src_v.size(); j += num_threads) {
          if (h_weights_v) {
            thread_edgelist.append(
              thread_handle, h_src_v[j], h_dst_v[j], (*h_weights_v)[j], std::nullopt, std::nullopt);
          } else {
            thread_edgelist.append(
              thread_handle, h_src_v[j], h_dst_v[j], std::nullopt, std::nullopt, std::nullopt);
          }
        }

        thread_edgelist.flush(thread_handle);
      });
    }

    // Wait for CPU threads to complete
    std::for_each(running_threads.begin(), running_threads.end(), [](auto& t) { t.join(); });
    running_threads.resize(0);
    instance_manager->reset_threads();

    std::cout << "now create Graph, num_gpus = " << num_gpus << std::endl;

    // TODO: At this point, the edgelist should be complete on the GPU.  We should be able to create
    // the graph
    //    Should test case where this loop runs more than num_gpus times and less.  Ideally, more
    //    should work fine, less should fail.
    for (int i = 0; i < num_gpus; ++i) {
      running_threads.emplace_back([&instance_manager,
                                    &graph,
                                    &graph_view,
                                    &edgelist,
                                    &renumber_map,
                                    &pageranks,
                                    is_symmetric = is_symmetric,
                                    renumber,
                                    do_expensive_check]() {
        auto thread_handle = instance_manager->get_handle();

        std::cout << "in threads, rank = " << thread_handle.get_rank()
                  << ", thread_rank = " << thread_handle.get_thread_rank() << std::endl;

        if (thread_handle.get_thread_rank() > 0) return;

        std::optional<cugraph::mtmg::edge_property_t<
          cugraph::mtmg::graph_view_t<vertex_t, edge_t, true, multi_gpu>,
          weight_t>>
          edge_weights{std::nullopt};
        std::optional<cugraph::mtmg::edge_property_t<
          cugraph::mtmg::graph_view_t<vertex_t, edge_t, true, multi_gpu>,
          edge_t>>
          edge_ids{std::nullopt};
        std::optional<cugraph::mtmg::edge_property_t<
          cugraph::mtmg::graph_view_t<vertex_t, edge_t, true, multi_gpu>,
          int32_t>>
          edge_types{std::nullopt};

        graph.initialize_pointer(thread_handle, thread_handle.raft_handle());

        edgelist.get_pointer(thread_handle)->finalize_buffer(thread_handle);
        edgelist.get_pointer(thread_handle)->consolidate_and_shuffle(thread_handle, true);

        raft::print_device_vector(" edgelist_majors",
                                  edgelist.get_pointer(thread_handle)->get_dst()[0].data(),
                                  edgelist.get_pointer(thread_handle)->get_dst()[0].size(),
                                  std::cout);
        raft::print_device_vector(" edgelist_minors",
                                  edgelist.get_pointer(thread_handle)->get_src()[0].data(),
                                  edgelist.get_pointer(thread_handle)->get_src()[0].size(),
                                  std::cout);

        cugraph::mtmg::
          create_graph_from_edgelist<vertex_t, edge_t, weight_t, edge_t, int32_t, true, multi_gpu>(
            thread_handle,
            edgelist,
            cugraph::graph_properties_t{is_symmetric, true},
            renumber,
            graph,
            edge_weights,
            edge_ids,
            edge_types,
            renumber_map,
            do_expensive_check);

        graph.set_view(thread_handle, graph_view);
        pageranks.initialize_pointer(
          thread_handle,
          graph_view.get_pointer(thread_handle)->local_vertex_partition_range_size(),
          thread_handle.raft_handle().get_stream());
      });
    }

    // Wait for CPU threads to complete
    std::for_each(running_threads.begin(), running_threads.end(), [](auto& t) { t.join(); });
    running_threads.resize(0);
    instance_manager->reset_threads();

    //   TODO: Try a facade for mtmg::pagerank
    //
    //    Should test case where this loop runs more than num_gpus times and less.  Ideally, more
    //    should work fine, less should fail.
    for (int i = 0; i < num_gpus; ++i) {
      running_threads.emplace_back([&instance_manager, &graph_view, &pageranks, alpha, epsilon]() {
        auto thread_handle = instance_manager->get_handle();

        if (thread_handle.get_thread_rank() > 0) return;

        // initialize to 0 for now
        auto& p = *pageranks.get_pointer(thread_handle);
        cugraph::detail::scalar_fill(thread_handle.raft_handle(), p.data(), p.size(), weight_t{0});
      });
    }

    // Wait for CPU threads to complete
    std::for_each(running_threads.begin(), running_threads.end(), [](auto& t) { t.join(); });
    running_threads.resize(0);
    instance_manager->reset_threads();

    std::cout << "compute pageranks..." << std::endl;

    std::vector<std::tuple<std::vector<vertex_t>, std::vector<result_t>>> computed_pageranks_v;
    std::mutex computed_pageranks_lock{};

    // Load computed_pageranks from different threads.  We'll use more threads than GPUs here
    // for (int i = 0; i < num_threads; ++i) {
    for (int i = 0; i < num_gpus; ++i) {
      running_threads.emplace_back([&instance_manager,
                                    &graph_view,
                                    &pageranks,
                                    &computed_pageranks_lock,
                                    &computed_pageranks_v,
                                    &h_src_v,
                                    &h_dst_v,
                                    &h_weights_v,
                                    i,
                                    num_threads]() {
        auto thread_handle = instance_manager->get_handle();

        auto number_of_vertices = graph_view.get_pointer(thread_handle)->number_of_vertices();

        std::cout << "number_of_vertices on rank " << thread_handle.get_rank() << " = "
                  << number_of_vertices << std::endl;

        std::vector<vertex_t> my_vertex_list;
        my_vertex_list.reserve((number_of_vertices + num_threads - 1) / num_threads);

        for (int j = i; j < number_of_vertices; j += num_threads) {
          my_vertex_list.push_back(j);
        }

        rmm::device_uvector<vertex_t> d_my_vertex_list(my_vertex_list.size(),
                                                       thread_handle.raft_handle().get_stream());
        raft::update_device(d_my_vertex_list.data(),
                            my_vertex_list.data(),
                            my_vertex_list.size(),
                            thread_handle.raft_handle().get_stream());

        sleep(thread_handle.get_rank());
        std::cout << "calling pageranks.gather, rank = " << thread_handle.get_rank() << std::endl;
        raft::print_device_vector("  pageranks",
                                  pageranks.get_pointer(thread_handle)->data(),
                                  pageranks.get_pointer(thread_handle)->size(),
                                  std::cout);
        auto d_my_pageranks = pageranks.gather(
          thread_handle,
          raft::device_span<vertex_t const>{d_my_vertex_list.data(), d_my_vertex_list.size()},
          graph_view);

        std::vector<result_t> my_pageranks(d_my_pageranks.size());
        raft::update_host(my_pageranks.data(),
                          d_my_pageranks.data(),
                          d_my_pageranks.size(),
                          thread_handle.raft_handle().get_stream());

        {
          std::lock_guard<std::mutex> lock(computed_pageranks_lock);
          raft::print_host_vector(
            "  my_vertex_list", my_vertex_list.data(), my_vertex_list.size(), std::cout);
          raft::print_host_vector(
            "  my_pageranks", my_pageranks.data(), my_pageranks.size(), std::cout);
          computed_pageranks_v.push_back(
            std::make_tuple(std::move(my_vertex_list), std::move(my_pageranks)));
        }
      });
    }

    // Wait for CPU threads to complete
    std::for_each(running_threads.begin(), running_threads.end(), [](auto& t) { t.join(); });
    running_threads.resize(0);
    instance_manager->reset_threads();
  }
};

using Tests_Multithreaded_File = Tests_Multithreaded<cugraph::test::File_Usecase>;
using Tests_Multithreaded_Rmat = Tests_Multithreaded<cugraph::test::Rmat_Usecase>;

// FIXME: add tests for type combinations
TEST_P(Tests_Multithreaded_File, CheckInt32Int32FloatFloat)
{
  run_current_test<int32_t, int32_t, float, float, true>(
    override_File_Usecase_with_cmd_line_arguments(GetParam()), std::vector<int>{{0, 1}});
}

TEST_P(Tests_Multithreaded_Rmat, CheckInt32Int32FloatFloat)
{
  run_current_test<int32_t, int32_t, float, float, true>(
    override_Rmat_Usecase_with_cmd_line_arguments(GetParam()), std::vector<int>{{0, 1}});
}

INSTANTIATE_TEST_SUITE_P(file_test,
                         Tests_Multithreaded_File,
                         ::testing::Combine(
                           // enable correctness checks
                           ::testing::Values(Multithreaded_Usecase{false, true},
                                             Multithreaded_Usecase{true, true}),
                           ::testing::Values(cugraph::test::File_Usecase("karate.csv"),
                                             cugraph::test::File_Usecase("dolphins.csv"))));

INSTANTIATE_TEST_SUITE_P(
  rmat_small_test,
  Tests_Multithreaded_Rmat,
  ::testing::Combine(
    // enable correctness checks
    ::testing::Values(Multithreaded_Usecase{true, false}, Multithreaded_Usecase{true, true}),
    ::testing::Values(cugraph::test::Rmat_Usecase(10, 16, 0.57, 0.19, 0.19, 0, false, false))));

INSTANTIATE_TEST_SUITE_P(
  file_benchmark_test, /* note that the test filename can be overridden in benchmarking (with
                          --gtest_filter to select only the file_benchmark_test with a specific
                          vertex & edge type combination) by command line arguments and do not
                          include more than one File_Usecase that differ only in filename
                          (to avoid running same benchmarks more than once) */
  Tests_Multithreaded_File,
  ::testing::Combine(
    // disable correctness checks
    ::testing::Values(Multithreaded_Usecase{false, false}, Multithreaded_Usecase{true, false}),
    ::testing::Values(cugraph::test::File_Usecase("test/datasets/karate.mtx"))));

INSTANTIATE_TEST_SUITE_P(
  rmat_benchmark_test, /* note that scale & edge factor can be overridden in benchmarking (with
                          --gtest_filter to select only the rmat_benchmark_test with a specific
                          vertex & edge type combination) by command line arguments and do not
                          include more than one Rmat_Usecase that differ only in scale or edge
                          factor (to avoid running same benchmarks more than once) */
  Tests_Multithreaded_Rmat,
  ::testing::Combine(
    // disable correctness checks for large graphs
    ::testing::Values(Multithreaded_Usecase{false, false}, Multithreaded_Usecase{true, false}),
    ::testing::Values(cugraph::test::Rmat_Usecase(10, 16, 0.57, 0.19, 0.19, 0, false, false))));

CUGRAPH_TEST_PROGRAM_MAIN()
